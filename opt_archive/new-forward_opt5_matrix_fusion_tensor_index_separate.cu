#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>
#include <mma.h>
using namespace nvcuda;

__global__ void mat_forward_kernel_tensor(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int b = blockIdx.z;
    int numAColumns = C*K*K;
    int numBColumns = H_out*W_out;

    __shared__ half tileA[16][16];
    __shared__ half tileB[16][16];
    __shared__ float tileC[16][16];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0);

    
    for (int i = 0; i < ceil(numAColumns/(float) 16); i++) {
        int ColA = i * 16 + threadIdx.x;
        int RowB = i * 16 + threadIdx.x;
        int c = RowB / (K*K);
        int p = RowB % (K*K) / K;   
        int q = RowB % K;
        for (int j = 0; j < 8; j++) {
            int RowA = blockIdx.y * 16 + threadIdx.y * 8 + j;
            int ColB = blockIdx.x * 16 + threadIdx.y * 8 + j;
            int h = ColB / W_out;
            int w = ColB % W_out;
            if (RowA < M && ColA < numAColumns)
                tileA[threadIdx.y * 8 + j][threadIdx.x] = __float2half(mask[RowA * numAColumns + ColA]);
            else
                tileA[threadIdx.y * 8 + j][threadIdx.x] = __float2half(0.0f);
            if (ColB < numBColumns && RowB < numAColumns)
                tileB[threadIdx.x][threadIdx.y * 8 + j] = __float2half(in_4d(b, c, h * S + p, w * S + q));
            else
                tileB[threadIdx.x][threadIdx.y * 8 + j] = __float2half(0.0f);
        }

        wmma::load_matrix_sync(a_frag, (half *) tileA, 16);
        wmma::load_matrix_sync(b_frag, (half *) tileB, 16);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    wmma::store_matrix_sync((float *) tileC, c_frag, 16, wmma::mem_row_major);
    for (int i = 0; i < 8; i++) {
        int RowC = blockIdx.y * 16 + threadIdx.y * 8 + i;
        int ColC = blockIdx.x * 16 + threadIdx.x;
        if (RowC < M && ColC < numBColumns)
            output[b * (M * numBColumns) + RowC * numBColumns + ColC] = tileC[threadIdx.y * 8 + i][threadIdx.x];
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

__global__ void mat_forward_kernel_tensor_8_32_16(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int b = blockIdx.z;
    int numAColumns = C*K*K;
    int numBColumns = H_out*W_out;

    __shared__ half tileA[8][16];
    __shared__ half tileB[16][32];
    __shared__ float tileC[8][32];
    __shared__ int c_arr[16];
    __shared__ int p_arr[16];
    __shared__ int q_arr[16];

    wmma::fragment<wmma::matrix_a, 8, 32, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 8, 32, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 8, 32, 16, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0);

    
    for (int i = 0; i < ceil(numAColumns/(float) 16); i++) {
        if (threadIdx.x < 16) {
            for (int j = 0; j < 8; j++) {
                int ColA = i * 16 + threadIdx.x;
                int RowA = blockIdx.y * 8 + j;
                if (RowA < M && ColA < numAColumns)
                    tileA[j][threadIdx.x] = __float2half(mask[RowA * numAColumns + ColA]);
                else
                    tileA[j][threadIdx.x] = __float2half(0.0f);
            }
            int RowB = i * 16 + threadIdx.x;
            c_arr[threadIdx.x] = RowB / (K*K);
            p_arr[threadIdx.x] = RowB % (K*K) / K;
            q_arr[threadIdx.x] = RowB % K;
        }
        int ColB = blockIdx.x * 32 + threadIdx.x;
        int h = ColB / W_out;
        int w = ColB % W_out;
        for (int j = 0; j < 16; j++) {
            int RowB = i * 16 + j;
            if (ColB < numBColumns && RowB < numAColumns)
                tileB[j][threadIdx.x] = __float2half(in_4d(b, c_arr[j], h * S + p_arr[j], w * S + q_arr[j]));
            else
                tileB[j][threadIdx.x] = __float2half(0.0f);
        }

        wmma::load_matrix_sync(a_frag, (half *) tileA, 16);
        wmma::load_matrix_sync(b_frag, (half *) tileB, 32);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    wmma::store_matrix_sync((float *) tileC, c_frag, 32, wmma::mem_row_major);
    for (int i = 0; i < 8; i++) { 
        int RowC = blockIdx.y * 8 + i;
        int ColC = blockIdx.x * 32 + threadIdx.x;
        if (RowC < M && ColC < numBColumns)
            output[b * (M * numBColumns) + RowC * numBColumns + ColC] = tileC[i][threadIdx.x];
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    hipMalloc((void **)device_input_ptr, B*C*H*W*sizeof(float));
    hipMalloc((void **)device_output_ptr, B*M*H_out*W_out*sizeof(float));
    hipMalloc((void **)device_mask_ptr, M*C*K*K*sizeof(float));


    hipMemcpy(*device_input_ptr, host_input, B*C*H*W*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, M*C*K*K*sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    if (M >= 16) {
        dim3 dimBlock = dim3(16, 2, 1);
        dim3 dimGrid = dim3(ceil((H_out*W_out)/(float)16), ceil(M/(float)16), B);
        mat_forward_kernel_tensor<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    } else {
        dim3 dimBlock = dim3(32, 1, 1);
        dim3 dimGrid = dim3(ceil((H_out*W_out)/(float)32), ceil(M/(float)8), B);
        mat_forward_kernel_tensor_8_32_16<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    }

    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // Copy the output back to host
    hipMemcpy(host_output, device_output, B*M*H_out*W_out*sizeof(float), hipMemcpyDeviceToHost);
   
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
