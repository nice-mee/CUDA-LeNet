#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>

#define TILE_WIDTH 16

__global__ void fp32_to_fp16(half *dst, const float *src, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        dst[i] = __float2half(src[i]);
    }
}

__global__ void fp16_to_fp32(float *dst, const half *src, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        dst[i] = __half2float(src[i]);
    }
}

__global__ void mat_forward_kernel(half *output, const half *input, const half *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int b = blockIdx.z;

    __shared__ half tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ half tileB[TILE_WIDTH][TILE_WIDTH];

    int numAColumns = C*K*K;
    int numBColumns = H_out*W_out;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    half Cvalue = 0.0;

    for (int i = 0; i < ceil(numAColumns/(float) TILE_WIDTH); i++) {
        int ColA = i * TILE_WIDTH + threadIdx.x;
        int RowB = i * TILE_WIDTH + threadIdx.y;
        int c = RowB / (K*K);
        int p = RowB % (K*K) / K;
        int q = RowB % (K*K) % K;
        int h = Col / W_out;
        int w = Col % W_out;
        if (Row < M && ColA < numAColumns)
            tileA[threadIdx.y][threadIdx.x] = mask[Row * numAColumns + ColA];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0;
        if (Col < numBColumns && RowB < numAColumns)
            tileB[threadIdx.y][threadIdx.x] = in_4d(b, c, h * S + p, w * S + q);
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0;
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; j++) {
            Cvalue = __hadd(Cvalue, __hmul(tileA[threadIdx.y][j], tileB[j][threadIdx.x]));
        }
        __syncthreads();
    }
    if (Row < M && Col < numBColumns)
        output[b * (M * numBColumns) + Row * numBColumns + Col] = Cvalue;

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    hipMalloc((void **)device_input_ptr, B*C*H*W*sizeof(float));
    hipMalloc((void **)device_output_ptr, B*M*H_out*W_out*sizeof(float));
    hipMalloc((void **)device_mask_ptr, M*C*K*K*sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, B*C*H*W*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, M*C*K*K*sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int W_size = ceil((float)W_out/TILE_WIDTH);
    int H_size = ceil((float)H_out/TILE_WIDTH);

    
    half *device_input_fp16;
    half *device_mask_fp16;
    half *device_output_fp16;
    hipMalloc((void **)&device_input_fp16, B*C*H*W*sizeof(half));
    hipMalloc((void **)&device_mask_fp16, M*C*K*K*sizeof(half));
    hipMalloc((void **)&device_output_fp16, B*M*H_out*W_out*sizeof(half));

    fp32_to_fp16<<<ceil((float)B*C*H*W/(float)256), 256>>>(device_input_fp16, device_input, B*C*H*W);
    fp32_to_fp16<<<ceil((float)M*C*K*K/(float)256), 256>>>(device_mask_fp16, device_mask, M*C*K*K);

    dim3 dimBlock = dim3(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid = dim3(ceil((H_out*W_out)/(float)TILE_WIDTH), ceil(M/(float)TILE_WIDTH), B);
    mat_forward_kernel<<<dimGrid, dimBlock>>>(device_output_fp16, device_input_fp16, device_mask_fp16, B, M, C, H, W, K, S);

    fp16_to_fp32<<<ceil((float)B*M*H_out*W_out/(float)256), 256>>>(device_output, device_output_fp16, B*M*H_out*W_out);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    hipFree(device_input_fp16);
    hipFree(device_mask_fp16);
    hipFree(device_output_fp16);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // Copy the output back to host
    hipMemcpy(host_output, device_output, B*M*H_out*W_out*sizeof(float), hipMemcpyDeviceToHost);
   
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
